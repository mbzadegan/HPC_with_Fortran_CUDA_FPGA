#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

// Simple 2D Jacobi kernel (fp64/fp32).
// For fp16 you can extend with __half intrinsics later.

template <typename T>
__global__ void jacobi2d(const T* __restrict__ in, T* __restrict__ out,
                         int N, int M)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > 0 && i < N-1 && j > 0 && j < M-1) {
        out[i*M + j] = (T)0.25 * ( in[(i-1)*M + j] + in[(i+1)*M + j]
                                 + in[i*M + (j-1)] + in[i*M + (j+1)] );
    }
}

template <typename T>
void run_jacobi(int N, int M, int iters)
{
    size_t bytes = N * M * sizeof(T);
    T *h_in = (T*)malloc(bytes);
    T *h_out = (T*)malloc(bytes);

    // init: zero, hot top edge = 1
    for (int i=0; i<N; i++) {
        for (int j=0; j<M; j++) {
            h_in[i*M+j] = 0;
        }
    }
    for (int j=0; j<M; j++) h_in[0*M+j] = 1;

    T *d_in, *d_out;
    hipMalloc(&d_in, bytes);
    hipMalloc(&d_out, bytes);
    hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice);

    dim3 block(16,16);
    dim3 grid((M+block.x-1)/block.x, (N+block.y-1)/block.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int t=0; t<iters; t++) {
        jacobi2d<T><<<grid,block>>>(d_in,d_out,N,M);
        std::swap(d_in,d_out);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms=0;
    hipEventElapsedTime(&ms,start,stop);

    double mlups = (double)(N-2)*(M-2)*iters / (ms*1e3);

    // Print CSV line: backend,precision,N,M,iters,runtime_ms,MLUPS,rel_error
    const char* prec = (sizeof(T)==8 ? "f64" : "f32");
    printf("cuda,%s,%d,%d,%d,%.3f,%.3f,%.4e\n",
           prec,N,M,iters,ms,mlups,0.0);

    hipMemcpy(h_out, d_in, bytes, hipMemcpyDeviceToHost);
    hipFree(d_in);
    hipFree(d_out);
    free(h_in);
    free(h_out);
}

int main(int argc, char** argv)
{
    if (argc < 5) {
        printf("Usage: %s N M ITERS PREC(f64|f32)\n", argv[0]);
        return 1;
    }
    int N = atoi(argv[1]);
    int M = atoi(argv[2]);
    int iters = atoi(argv[3]);
    std::string prec(argv[4]);

    if (prec=="f64") run_jacobi<double>(N,M,iters);
    else if (prec=="f32") run_jacobi<float>(N,M,iters);
    else {
        fprintf(stderr,"Precision %s not implemented yet.\n", prec.c_str());
        return 1;
    }
    return 0;
}
